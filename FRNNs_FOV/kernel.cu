#include "hip/hip_runtime.h"
/**
*	Based off earlier start from:
*	https://github.com/Robadob/SP-Bench/commit/35dcbb81cc0b73cdb6b08fb622f13e688a878133
*	This technique only concerns 2D FoV limitation
*	In particular as may be applied to pedestrian models, whereby a pedestrian is assumed unable to see behind them.
*	We attempt to utilise a look-up table (similar to marching cubes), to optimise calculation of bins which intersect the given FoV
*/
#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include <glm/glm.hpp>
#include <glm/gtx/component_wise.hpp>
#include <glm/gtc/constants.hpp>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hipcub/hipcub.hpp>
#include <glm/gtc/epsilon.hpp>
#include <glm/gtx/vector_angle.hpp>
#define EPSILON 0.005f
//#define CIRCLES
//Cuda call
static void HandleCUDAError(const char *file,
	int line,
	hipError_t status = hipGetLastError()) {
#ifdef _DEBUG
	hipDeviceSynchronize();
#endif
	if (status != hipError_t::hipSuccess || (status = hipGetLastError()) != hipError_t::hipSuccess)
	{
		printf("%s(%i) CUDA Error Occurred;\n%s\n", file, line, hipGetErrorString(status));
#ifdef _DEBUG
		getchar();
#endif
		exit(1);
	}
}
#define CUDA_CALL( err ) (HandleCUDAError(__FILE__, __LINE__ , err))
#define CUDA_CHECK() (HandleCUDAError(__FILE__, __LINE__))

//Logging (found in log.cpp)
#include <fstream>
void createLog(std::ofstream &f);
void log(std::ofstream &f,
	const unsigned int &estRadialNeighbours,
	const unsigned int &agentCount,
	const unsigned int &envWidth,
	const float &PBM_control,
	const float &kernel_control,
	const float &PBM,
	const float &kernel,
	const unsigned int &fails
);
__device__ __constant__ unsigned int d_agentCount;
__device__ __constant__ float d_environmentWidth_float;
__device__ __constant__ unsigned int d_gridDim;
glm::uvec2 GRID_DIMS;
__device__ __constant__ float d_gridDim_float;
__device__ __constant__ float d_RADIUS;
__device__ __constant__ float d_R_SIN_45;
__device__ __constant__ float d_binWidth;

//For thread block max bin check
unsigned int *d_PBM_max_count;
unsigned int PBM_max_count = 0;
unsigned int PBM_max_Moore_count = 0;//This is unused, it could be used if we wished to load entire Moore neighbourhood at once to shared mem, instead we load a bin at a time

texture<float4> d_texMessages;//float2 pos, float2 velocity
texture<unsigned int> d_texPBM;

__global__ void init_curand(hiprandState *state, unsigned long long seed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < d_agentCount)
		hiprand_init(seed, id, 0, &state[id]);
}
__global__ void init_agents(hiprandState *state, glm::vec4 *locationMessages) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= d_agentCount)
		return;
	//Position
	//curand_unform returns 0<x<=1.0, not much can really do about 0 exclusive
	//negate and  + 1.0, to make  0<=x<1.0
	locationMessages[id].x = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentWidth_float;
	locationMessages[id].y = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentWidth_float;
	//Velocity
	glm::vec2 vel = normalize(glm::vec2((hiprand_uniform(&state[id])-0.5)*2.0f, (hiprand_uniform(&state[id])-0.5f)*2.0f));
	locationMessages[id].z = vel.x;
	locationMessages[id].w = vel.y;
}
__device__ __forceinline__ glm::ivec2 getGridPosition(glm::vec2 worldPos)
{
	//Clamp each grid coord to 0<=x<dim
	return clamp(floor((worldPos / d_environmentWidth_float)*d_gridDim_float), glm::vec2(0), glm::vec2((float)d_gridDim - 1));
}
__device__ __forceinline__ unsigned int getHash(glm::ivec2 gridPos)
{
	//Bound gridPos to gridDimensions
	gridPos = clamp(gridPos, glm::ivec2(0), glm::ivec2(d_gridDim - 1));
	//Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
	return (unsigned int)(
		(gridPos.y * d_gridDim) +					//y
		gridPos.x); 	                            //x
}
__global__ void atomicHistogram(unsigned int* bin_index, unsigned int* bin_sub_index, unsigned int *pbm_counts, glm::vec4 *messageBuffer)
{
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	glm::ivec2 gridPos = getGridPosition(glm::vec2(messageBuffer[index].x, messageBuffer[index].y));
	unsigned int hash = getHash(gridPos);
	bin_index[index] = hash;
	unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
	bin_sub_index[index] = bin_idx;
}
__global__ void reorderLocationMessages(
	unsigned int* bin_index,
	unsigned int* bin_sub_index,
	unsigned int *pbm,
	glm::vec4 *unordered_messages,
	glm::vec4 *ordered_messages
)
{
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	unsigned int i = bin_index[index];
	unsigned int sorted_index = pbm[i] + bin_sub_index[index];

	//Order messages into swap space
	ordered_messages[sorted_index] = unordered_messages[index];
}
int requiredSM(int blockSize)
{
	hipDeviceProp_t dp;
	int device;
	hipGetDevice(&device);
	memset(&dp, sizeof(hipDeviceProp_t), 0);
	hipGetDeviceProperties(&dp, device);
	//We could use dp.sharedMemPerBlock/N to improve occupancy
	return (int)min(PBM_max_count * sizeof(float2), dp.sharedMemPerBlock);//Need to limit this to the max SM
}

__forceinline__ __device__ void avoidSum(const glm::vec2 &mePos, const glm::vec2 &meVec, const glm::vec2 &msgPos, const glm::vec2 &msgVec, glm::vec2 &nVel, glm::vec2 &aVel)
{
#define SPEED_LIMIT 1.0f
#define TIME_SCALER	0.0003f
#define MIN_DISTANCE 0.0001f
#define SCALE_FACTOR 0.03125
#define I_SCALER (SCALE_FACTOR*0.35f)
#define STEER_WEIGHT		0.10f
#define AVOID_WEIGHT		0.02f
#define COLLISION_WEIGHT	0.50f
#define GOAL_WEIGHT			0.20f
	//Lightweight bounds check
	glm::vec2 offset = msgPos - mePos;
	float distance = glm::length(offset);
	if (distance <d_RADIUS && distance > MIN_DISTANCE)
	{
		//FOV Check
		float angle = glm::angle(meVec, offset);
		if (angle<1.5708)//d_HALF_FOV (90 degrees in radians)
		{
			float perception = 45.0f;
			//STEER
			if ((angle < glm::radians(perception)) || (angle > 3.14159265f - glm::radians(perception))) {
				glm::vec2 s_velocity = -offset;
				s_velocity *= powf(I_SCALER / distance, 1.25f)*STEER_WEIGHT;
				nVel += s_velocity;
			}

			//AVOID
			glm::vec2 a_velocity = -offset;
			a_velocity *= powf(I_SCALER / distance, 2.00f)*AVOID_WEIGHT;
			aVel += a_velocity;
		}
	}

}
/**
* Kernel must be launched 1 block per bin
* This removes the necessity of __launch_bounds__(64) as all threads in block are touching the same messages
* However we end up with alot of (mostly) idle threads if one bin dense, others empty.
*/
__global__  void __launch_bounds__(64) neighbourSearch_control(const glm::vec4 *agents, glm::vec4 *out)
{
#define STRIPS
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;
	glm::vec2 pos = glm::vec2(agents[index].x, agents[index].y);
	glm::vec2 vel = glm::vec2(agents[index].z, agents[index].w);
	glm::vec2 navigate_velocity = glm::vec2(0);
	glm::vec2 avoid_velocity = glm::vec2(0);
	glm::ivec2 gridPos = getGridPosition(pos);
	glm::ivec2 gridPosRelative;

	for (gridPosRelative.y = -1; gridPosRelative.y <= 1; gridPosRelative.y++)
	{//ymin to ymax
		int currentBinY = gridPos.y + gridPosRelative.y;
		if (currentBinY >= 0 && currentBinY < d_gridDim)
		{
#ifndef STRIPS
			for (gridPosRelative.x = -1; gridPosRelative.x <= 1; gridPosRelative.x++)
			{//xmin to xmax
				int currentBinX = gridPos.x + gridPosRelative.x;
				//Find bin start and end
				unsigned int binHash = getHash(glm::ivec2(currentBinX, currentBinY));
				//if (binHash>d_gridDim*d_gridDim)
				//{
				//    printf("Hash: %d, gridDim: %d, pos: (%d, %d)\n", binHash, d_gridDim, tGridPos.x, tGridPos.y);
				//}
				unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
				unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
#else

			int currentBinX = gridPos.x - 1;
			currentBinX = currentBinX >= 0 ? currentBinX : 0;
			unsigned int binHash = getHash(glm::ivec2(currentBinX, currentBinY));
			unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
			currentBinX = gridPos.x + 1;
			currentBinX = currentBinX < d_gridDim ? currentBinX : d_gridDim - 1;
			binHash = getHash(glm::ivec2(currentBinX, currentBinY));
			unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
#endif
			//Iterate messages in range
			for (unsigned int i = binStart; i < binEnd; ++i)
			{
				//if (i != index)//Ignore self
				{
					float4 message = tex1Dfetch(d_texMessages, i);
					glm::vec2 *_pos = (glm::vec2*)&message;
					glm::vec2 *_vel = (glm::vec2*)&(message.z);

					avoidSum(pos, vel, *_pos, *_vel, navigate_velocity, avoid_velocity);
				}
			}
			}
#ifndef STRIPS
		}
#endif
	}

	//Process result of avoidsum
	{
		//random walk goal
		glm::vec2 goal_velocity = vel * GOAL_WEIGHT;

		//maximum velocity rule
		goal_velocity += navigate_velocity + avoid_velocity;

		float current_speed = length(vel) + 0.025f;
		vel += current_speed * goal_velocity;
		float speed = length(vel);
		//limit speed
		if (speed >= SPEED_LIMIT) {
			vel = normalize(vel)*SPEED_LIMIT;
			speed = SPEED_LIMIT;
		}

		//update position
		pos += vel*TIME_SCALER;
	}

out[index] = glm::vec4(pos, vel);
}
/**
* Kernel must be launched 1 block per bin
* This removes the necessity of __launch_bounds__(64) as all threads in block are touching the same messages
* However we end up with alot of (mostly) idle threads if one bin dense, others empty.
*/
__global__ void neighbourSearch(const glm::vec4 *agents, glm::vec4 *out)
{
	glm::ivec2 relatives[8] = { 
		glm::ivec2(0, 1),	//North
		glm::ivec2(1,1),	//North East
		glm::ivec2(1,0),    //East
		glm::ivec2(1, -1),	//South East
		glm::ivec2(0, -1),	//South
		glm::ivec2(-1, -1), //South West
		glm::ivec2(-1, 0),	//West
		glm::ivec2(-1, 1)	//North West
	};
	enum Quadrant {NW, NE, SW, SE};

	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	//My data
	glm::vec2 navigate_velocity = glm::vec2(0.0f, 0.0f);
	glm::vec2 avoid_velocity = glm::vec2(0.0f, 0.0f);
	int __relativeIndex;
	unsigned int __relativeCount;
	glm::vec2 pos, vel;
	{
		//Load self
		pos = glm::vec2(agents[index].x, agents[index].y);
		vel = glm::vec2(agents[index].z, agents[index].w);
	}
	glm::ivec2 myBin = getGridPosition(pos);
	{
		//Process relative (0, 0)
		unsigned int binHash = getHash(myBin);
		unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
		unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
		//unsigned int binCount = binEnd - binStart;
		for (unsigned int j = binStart; j<binEnd; ++j)
		{
			if (j != index)
			{
				float4 message = tex1Dfetch(d_texMessages, binStart + threadIdx.x);
				glm::vec2 _pos = glm::vec2(message.x, message.y);
				glm::vec2 _vel = glm::vec2(message.z, message.w);
				avoidSum(pos, vel, _pos, _vel, navigate_velocity, avoid_velocity);
			}
		}
	}
	//Identify the relative element which contains dir
	{
		//incremenet pos by vel * unit
		glm::vec2 dest = pos + (glm::normalize(vel)*d_binWidth);
		//Find which bin this resides in
		glm::ivec2 destBin = getGridPosition(dest);
		//Convert this bin to a relative index
		glm::ivec2 destOffset = myBin - destBin;
		assert(destOffset != glm::ivec2(0));
		//Identify index where that falls in 'relatives' array
		if (destOffset.x == 1)
		{
			__relativeIndex = 2 - destOffset.y;
		}
		else if (destOffset.x == -1)
		{
			__relativeIndex = 6 - destOffset.y;
		}
		else
		{
			__relativeIndex = 2 - 2*destOffset.y;
		}
	//Rotate about circle -FOV/2 (how many elements is this?
		//180 degrees requires 2 on either side of central
		__relativeIndex -= 2;
		__relativeCount = 5;
		//
		glm::vec2 qPos = pos - glm::vec2(glm::ivec2(pos));//Just want the decimal part
		if (qPos.x > 0)qPos.x = 1;
		else if(qPos.x < 0)qPos.x = -1;
		if (qPos.y > 0)qPos.y = 1;
		else if (qPos.y < 0)qPos.y = -1;
		glm::ivec2 _qPos = qPos;
		//+-1 on either side, based on the quadrant relative to velocity
		//Temp(?) max all
		__relativeIndex -= 1;
		__relativeCount += 2;
		//Correct for overflow
		__relativeIndex = (__relativeIndex + 8) % 8;//+8 to account for underflow (% is remainder op, not mod)
	}
	//Iterate FOV relatives across
	for(unsigned int i = 0;i<__relativeCount;++i)
	{
		unsigned int currentIndex = __relativeIndex + i;
		currentIndex = currentIndex >= 8 ? currentIndex - 8 : currentIndex;//(__relativeIndex+i)%8
		glm::ivec2 currentBin = myBin + relatives[currentIndex];
		if (currentBin.x >= 0 && currentBin.x < d_gridDim)
		{
			if (currentBin.y >= 0 && currentBin.y < d_gridDim)
			{
				//Now we must load all messages from currentBin
				unsigned int binHash = getHash(currentBin);
				unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
				unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
				for(unsigned int j = binStart;j<binEnd;++j)
				{
					float4 message = tex1Dfetch(d_texMessages, binStart + threadIdx.x);
					glm::vec2 _pos = glm::vec2(message.x, message.y);
					glm::vec2 _vel = glm::vec2(message.z, message.w);
					avoidSum(pos, vel, _pos, _vel, navigate_velocity, avoid_velocity);
				}
			}
		}
				
	}

	//Process result of avoidsum
	{
		//random walk goal
		glm::vec2 goal_velocity = vel * GOAL_WEIGHT;

		//maximum velocity rule
		goal_velocity += navigate_velocity + avoid_velocity;

		float current_speed = length(vel) + 0.025f;
		vel += current_speed * goal_velocity;
		float speed = length(vel);
		//limit speed
		if (speed >= SPEED_LIMIT) {
			vel = normalize(vel)*SPEED_LIMIT;
			speed = SPEED_LIMIT;
		}

		//update position
		pos += vel*TIME_SCALER;
	}
	
	
	//Output
	out[index] = glm::vec4(pos, vel);
}


__global__ void unsortMessages(
	unsigned int* bin_index,
	unsigned int* bin_sub_index,
	unsigned int *pbm,
	glm::vec4 *ordered_messages,
	glm::vec4 *unordered_messages
)
{
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	unsigned int i = bin_index[index];
	unsigned int sorted_index = pbm[i] + bin_sub_index[index];

	//Order messages into swap space
	unordered_messages[index] = ordered_messages[sorted_index];
}
/**
* This program is to act as a test rig to demonstrate the raw impact of raw message handling
*/
void run(std::ofstream &f, const unsigned int ENV_WIDTH, const unsigned int AGENT_COUNT = 1000000)
{
	void *d_CUB_temp_storage = nullptr;
	size_t d_CUB_temp_storage_bytes = 0;
	//Spatial partitioning mock
	//Fixed 2D environment of 1000x1000
	//Filled with 1,000,000 randomly distributed agents
	//const unsigned int ENV_WIDTH = 250;
	float ENV_WIDTH_float = (float)ENV_WIDTH;
	const unsigned int RNG_SEED = 12;
	const unsigned int ENV_VOLUME = ENV_WIDTH * ENV_WIDTH;
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_agentCount), &AGENT_COUNT, sizeof(unsigned int)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentWidth_float), &ENV_WIDTH_float, sizeof(float)));
	glm::vec4 *d_agents_init = nullptr, *d_agents = nullptr, *d_out = nullptr;
	unsigned int *d_keys = nullptr, *d_vals = nullptr;
	CUDA_CALL(hipMalloc(&d_agents_init, sizeof(glm::vec4) * AGENT_COUNT));
	CUDA_CALL(hipMalloc(&d_agents, sizeof(glm::vec4) * AGENT_COUNT));
	CUDA_CALL(hipMalloc(&d_out, sizeof(glm::vec4) * AGENT_COUNT));
	glm::vec2 *h_out = (glm::vec2*)malloc(sizeof(glm::vec4) * AGENT_COUNT);
	glm::vec2 *h_out_control = (glm::vec2*)malloc(sizeof(glm::vec4) * AGENT_COUNT);
	//Init agents
	{
		//Generate hiprand
		hiprandState *d_rng;
		CUDA_CALL(hipMalloc(&d_rng, AGENT_COUNT * sizeof(hiprandState)));
		//Arbitrary thread block sizes (speed not too important during one off initialisation)
		unsigned int initThreads = 512;
		unsigned int initBlocks = (AGENT_COUNT / initThreads) + 1;
		init_curand << <initBlocks, initThreads >> >(d_rng, RNG_SEED);//Defined in CircleKernels.cuh
		CUDA_CALL(hipDeviceSynchronize());
		init_agents << <initBlocks, initThreads >> >(d_rng, d_agents_init);
		//Free hiprand
		CUDA_CALL(hipFree(d_rng));
		CUDA_CALL(hipMalloc(&d_keys, sizeof(unsigned int)*AGENT_COUNT));
		CUDA_CALL(hipMalloc(&d_vals, sizeof(unsigned int)*AGENT_COUNT));
	}
	//Decide interaction radius
	//for a range of bin widths
	const float RADIUS = 1.0f;//
	const float RADIAL_VOLUME = glm::pi<float>()*RADIUS*RADIUS;
	const unsigned int AVERAGE_NEIGHBOURS = (unsigned int)(AGENT_COUNT*RADIAL_VOLUME / ENV_VOLUME);
	printf("Agents: %d, RVol: %.2f, Average Neighbours: %d\n", AGENT_COUNT, RADIAL_VOLUME, AVERAGE_NEIGHBOURS);
	//{
	//    hipFree(d_agents_init);
	//    hipFree(d_agents);
	//    hipFree(d_out);
	//    return;
	//}

	const float rSin45 = (float)(RADIUS*sin(glm::radians(45)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_RADIUS), &RADIUS, sizeof(float)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_R_SIN_45), &rSin45, sizeof(float)));
	{
		{
			//Copy init state to d_out   
			CUDA_CALL(hipMemcpy(d_out, d_agents_init, sizeof(glm::vec4)*AGENT_COUNT, hipMemcpyDeviceToDevice));
		}
		//Decide bin width (as a ratio to radius)
		const float BIN_WIDTH = RADIUS;
		float GRID_DIMS_float = ENV_WIDTH / BIN_WIDTH;
		GRID_DIMS = glm::uvec2((unsigned int)ceil(GRID_DIMS_float));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_binWidth), &BIN_WIDTH, sizeof(float)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim), &GRID_DIMS.x, sizeof(unsigned int)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim_float), &GRID_DIMS_float, sizeof(float)));
		const unsigned int BIN_COUNT = glm::compMul(GRID_DIMS);
		hipEvent_t start_PBM, end_PBM, start_kernel, end_kernel;
		hipEventCreate(&start_PBM);
		hipEventCreate(&end_PBM);
		hipEventCreate(&start_kernel);
		hipEventCreate(&end_kernel);
		//BuildPBM
		unsigned int *d_PBM_counts = nullptr;
		unsigned int *d_PBM = nullptr;
		CUDA_CALL(hipMalloc(&d_PBM_counts, (BIN_COUNT + 1) * sizeof(unsigned int)));
		CUDA_CALL(hipMalloc(&d_PBM, (BIN_COUNT + 1) * sizeof(unsigned int)));
		//Prep for threadblocks
		CUDA_CALL(hipMalloc(&d_PBM_max_count, sizeof(unsigned int)));
		CUDA_CALL(hipMemset(d_PBM_max_count, 0, sizeof(unsigned int)));
		{//Resize cub temp if required
			size_t bytesCheck, bytesCheck2;
			hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, d_PBM, d_PBM_counts, BIN_COUNT + 1);
			hipcub::DeviceReduce::Max(nullptr, bytesCheck2, d_PBM_counts, d_PBM_max_count, BIN_COUNT);
			bytesCheck = glm::max(bytesCheck, bytesCheck2);
			if (bytesCheck > d_CUB_temp_storage_bytes)
			{
				if (d_CUB_temp_storage)
				{
					CUDA_CALL(hipFree(d_CUB_temp_storage));
				}
				d_CUB_temp_storage_bytes = bytesCheck;
				CUDA_CALL(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
			}
		}

		float pbmMillis_control = 0, kernelMillis_control = 0;
		float pbmMillis = 0, kernelMillis = 0;
		for (unsigned int _j = 1; _j < UINT_MAX; --_j)
		{
			//1 = control
			//0 = threadblock
			bool isControl = _j != 0;

			//For 200 iterations (to produce an average)
			const unsigned int ITERATIONS = 1;
			for (unsigned int i = 0; i < ITERATIONS; ++i)
			{
				//Reset each run of average model
#ifndef CIRCLES
				CUDA_CALL(hipMemcpy(d_out, d_agents_init, sizeof(glm::vec4)*AGENT_COUNT, hipMemcpyDeviceToDevice));
#endif	
				hipEventRecord(start_PBM);
				{//Build atomic histogram
					CUDA_CALL(hipMemset(d_PBM_counts, 0x00000000, (BIN_COUNT + 1) * sizeof(unsigned int)));
					int blockSize;   // The launch configurator returned block size 
					CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram, 32, 0));//Randomly 32
																												 // Round up according to array size
					int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
					atomicHistogram << <gridSize, blockSize >> > (d_keys, d_vals, d_PBM_counts, d_out);
					CUDA_CALL(hipDeviceSynchronize());
				}
				{//Scan (sum), to finalise PBM
					hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_PBM_counts, d_PBM, BIN_COUNT + 1);
				}
				{//Reorder messages
					int blockSize;   // The launch configurator returned block size 
					CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, reorderLocationMessages, 32, 0));//Randomly 32
																														 // Round up according to array size
					int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
					//Copy messages from d_messages to d_messages_swap, in hash order
					reorderLocationMessages << <gridSize, blockSize >> > (d_keys, d_vals, d_PBM, d_out, d_agents);
					CUDA_CHECK();
				}
				if (!isControl)
				{//Calc max bin size (for threadblocks)
					hipcub::DeviceReduce::Max(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_PBM_counts, d_PBM_max_count, BIN_COUNT);
					CUDA_CALL(hipGetLastError());
					CUDA_CALL(hipMemcpy(&PBM_max_count, d_PBM_max_count, sizeof(unsigned int), hipMemcpyDeviceToHost));
					//Calc moore size (bin size^dims?)
					//PBM_max_Moore_count = (unsigned int)pow(PBM_max_count, 2);//2==2D//Unused, requires 9x shared mem in 2D, 27x in 3D
				}
				{//Fill PBM and Message Texture Buffers																			  
					CUDA_CALL(hipDeviceSynchronize());//Wait for return
					CUDA_CALL(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec4) * AGENT_COUNT));
					CUDA_CALL(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (BIN_COUNT + 1)));
				}
				hipEventRecord(end_PBM);
				hipEventRecord(start_kernel);
				if (isControl)
				{
					//Each message samples radial neighbours (static model)
					int blockSize;   // The launch configurator returned block size 
					CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, reorderLocationMessages, 32, 0));//Randomly 32
																														 // Round up according to array size
					int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
					//Copy messages from d_agents to d_out, in hash order
					neighbourSearch_control << <gridSize, blockSize >> > (d_agents, d_out);
					CUDA_CHECK();
				}
				else
				{
					//Each message samples radial neighbours (static model)
					int blockSize = PBM_max_count;   //blockSize == largest bin size
					dim3 gridSize;
					gridSize.x = GRID_DIMS.x;
					gridSize.y = GRID_DIMS.y;
					gridSize.z = 1;// GRID_DIMS.z;
								   //Copy messages from d_agents to d_out, in hash order
					neighbourSearch << <gridSize, blockSize, requiredSM(blockSize) >> > (d_agents, d_out);
					CUDA_CHECK();
				}
				CUDA_CALL(hipDeviceSynchronize());
				hipEventRecord(end_kernel);
				hipEventSynchronize(end_kernel);

				float _pbmMillis = 0, _kernelMillis = 0;
				hipEventElapsedTime(&_pbmMillis, start_PBM, end_PBM);
				hipEventElapsedTime(&_kernelMillis, start_kernel, end_kernel);
				if (isControl)
				{
					pbmMillis_control += _pbmMillis;
					kernelMillis_control += _kernelMillis;
				}
				else
				{
					pbmMillis += _pbmMillis;
					kernelMillis += _kernelMillis;
				}

			}//for(ITERATIONS)
			pbmMillis_control /= ITERATIONS;
			kernelMillis_control /= ITERATIONS;
			pbmMillis /= ITERATIONS;
			kernelMillis /= ITERATIONS;

			{//Unorder messages
				int blockSize;   // The launch configurator returned block size 
				CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, reorderLocationMessages, 32, 0));//Randomly 32
																													 // Round up according to array size
				int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
				//Copy messages from d_out to d_agents, in hash order
				unsortMessages << <gridSize, blockSize >> > (d_keys, d_vals, d_PBM, d_out, d_agents);
				CUDA_CHECK();
				//Swap d_out and d_agents
				{
					glm::vec4 *t = d_out;
					d_out = d_agents;
					d_agents = t;
				}
				//Wait for return
				CUDA_CALL(hipDeviceSynchronize());
				//Copy back to relative host array (for validation)
				CUDA_CALL(hipMemcpy(isControl ? h_out_control : h_out, d_out, sizeof(glm::vec4)*AGENT_COUNT, hipMemcpyDeviceToHost));
				CUDA_CALL(hipDeviceSynchronize());
			}
		}//for(MODE)
		CUDA_CALL(hipUnbindTexture(d_texPBM));
		CUDA_CALL(hipUnbindTexture(d_texMessages));
		CUDA_CALL(hipFree(d_PBM_counts));
		CUDA_CALL(hipFree(d_PBM));
		//log();
		printf("Control:     PBM: %.2fms, Kernel: %.2fms\n", pbmMillis_control, kernelMillis_control);
		printf("ThreadBlock: PBM: %.2fms, Kernel: %.2fms\n", pbmMillis, kernelMillis);
		unsigned int fails = 0;
#ifndef CIRCLES

		{//Validation
		 //Validate results for average model
		 //thrust::sort(thrust::cuda::par, d_out, d_out + AGENT_COUNT, vec2Compare());
		 //CUDA_CALL(hipMemcpy(isControl ? h_out_control : h_out, d_out, sizeof(glm::vec2)*AGENT_COUNT, hipMemcpyDeviceToHost));
			for (unsigned int i = 0; i < AGENT_COUNT; ++i)
			{
				assert(!(isnan(h_out[i].x) || isnan(h_out[i].y)));
				if (isnan(h_out[i].x) || isnan(h_out[i].y))
					printf("err nan\n");
				auto ret = glm::epsilonEqual(h_out[i], h_out_control[i], EPSILON);
				if (!(ret.x&&ret.y))
				{
					if (fails == 0)
						printf("(%.5f, %.5f) vs (%.5f, %.5f)\n", h_out_control[i].x, h_out_control[i].y, h_out[i].x, h_out[i].y);
					fails++;
				}
			}
			if (fails > 0)
				printf("%d/%d (%.1f%%) Failed.\n", fails, AGENT_COUNT, 100 * (fails / (float)AGENT_COUNT));
		}
#endif
		log(f, AVERAGE_NEIGHBOURS, AGENT_COUNT, ENV_WIDTH, pbmMillis_control, kernelMillis_control, pbmMillis, kernelMillis, fails);
	}

	CUDA_CALL(hipUnbindTexture(d_texMessages));
	CUDA_CALL(hipFree(d_vals));
	CUDA_CALL(hipFree(d_keys));
	CUDA_CALL(hipFree(d_agents));
	CUDA_CALL(hipFree(d_agents_init));
	CUDA_CALL(hipFree(d_out));
	free(h_out);
	free(h_out_control);
}
void runAgents(std::ofstream &f, const unsigned int AGENT_COUNT, const float DENSITY)
{
	//density refers to approximate number of neighbours
	run(f, (unsigned int)sqrt(AGENT_COUNT / (DENSITY*2.86 / 9)), AGENT_COUNT);
}
int main()
{
	{
		std::ofstream f;
		createLog(f);
		assert(f.is_open());
		for (unsigned int i = 20000; i <= 3000000; i += 20000)
		{
			//Run i agents in a density with roughly 60 radial neighbours, and log
			//Within this, it is tested over a range of proportional bin widths
			runAgents(f, i, 20);
			break;
		}
	}
	printf("fin\n");
	getchar();
	return 0;
}

